
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>

__global__ void checkIndex(void)
{
    printf("threadIdx:(%d, %d, %d)  blockIdx:(%d, %d, %d)  blockDim:(%d, %d, %d) "
           "gridDim:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
           gridDim.x,gridDim.y,gridDim.z);
}

int main()
{
    std::cout << "Hello, World!" << std::endl;

    int nElem = 5;

    dim3 block(3);
    dim3 grid((nElem+block.x-1)/block.x);

    // check grid and block dimension from host side
    printf("grid.x %d grid.y %d grid.z %d\n",grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x, block.y, block.z);

    // check grid and block dimension from device side
    checkIndex <<<grid, block>>> ();

    hipDeviceSynchronize();


    return 0;
}
