#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "learnCuda.h"


void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; ++iy)
    {
        for (int ix = 0; ix < nx; ++ix)
        {
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx;
        ib += nx;
        ic += nx;
    }
}

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
    // set up date size of vectors
    int nx = 1 << 14;
    int ny = 1 << 14;
    int nxy = nx * ny;
    printf("Matrix size:nx %d ny %d\n", nx, ny);

    // malloc host memory
    size_t nBytes = nxy * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *) malloc(nBytes);
    h_B = (float *) malloc(nBytes);
    hostRef = (float *) malloc(nBytes);
    gpuRef = (float *) malloc(nBytes);
    double iStart, iElaps;
    // initialize data at host side
    iStart = cpuSecond();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    iElaps = cpuSecond() - iStart;
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    // add vector at host side for result checks
    iStart = cpuSecond();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnHost Time elapsed %f" \
    "sec\n", iElaps);
    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float **) &d_A, nBytes);
    hipMalloc((float **) &d_B, nBytes);
    hipMalloc((float **) &d_C, nBytes);
    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    // invoke kernel at host side
    int dimx = 32;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    iStart = cpuSecond();
    sumMatrixOnGPU2D<<<grid, block>>>(d_A, d_B, d_C, nx, ny);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnGPU <<<(%d,%d), (%d,%d)>>> Time elapsed %f" \
    "sec\n", grid.x, grid.y, block.x, block.y, iElaps);
    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    // check device results
    checkResult(hostRef, gpuRef, nxy);
    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    return (0);
}
