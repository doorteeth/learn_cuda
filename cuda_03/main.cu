
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned int ) time(&t));

    for (int i = 0; i < size; ++i)
    {
        ip[i]=(float )(rand()&0xFF)/10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i < N; ++i)
    {
        C[i]=A[i]+B[i];
    }
}

__global__ void sumArraysOnDevice(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i < N; ++i)
    {
        C[i]=A[i]+B[i];
    }
}

int main()
{
    std::cout << "Hello, World!" << std::endl;

    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);


    float *h_A, *h_B, *h_C, *h_out;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    h_C = (float *)malloc(nBytes);
    h_out = (float *)malloc(nBytes);

    float *d_A, *d_B, *d_C;

    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);
//
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    sumArraysOnHost(h_A, h_B, h_C, nElem);
    sumArraysOnDevice<<<1,1>>>(d_A, d_B, d_C, nElem);


    hipDeviceSynchronize();
    hipMemcpy(h_out, d_C, nBytes, hipMemcpyDeviceToHost);


    double epsilon=1.0E-8;

    for (int i = 0; i < nElem; ++i)
    {
        if (abs(h_out[i]-h_C[i])>epsilon)
        {
            std::cout<<"Failure"<<std::endl;
            return 1;
        }
    }
    std::cout<<"Success"<<std::endl;

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_out);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
